#include "NewtonStepperCusp.h"

// general cuda includes
#include "hip/hip_runtime.h"
#include ""

// cusp and thurst includes
#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/zip_iterator.h>
#include <cusp/krylov/cg.h>
#include <cusp/monitor.h>


std::vector<float> NewtonStepperCusp::step(std::vector<int> I, std::vector<int> J, std::vector<float> V, std::vector<float> force)
{
	return std::vector<float>();
}

//NewtonMethodStepper::NewtonMethodStepper(ElementMesh * mesh) : BaseStepper(mesh)
//{
	/*
	totalExternalForce = Eigen::Vector3f::Zero();
	Eigen::Vector3f force(0,-0.1,0);
	mesh->externalForcesPerVertex.push_back(force);
	for (int i = 0; i < mesh->externalForcesPerVertex.size(); ++i)
	{
		totalExternalForce += mesh->externalForcesPerVertex[i];
	}
	*/
//}


//void NewtonMethodStepper::step()
//{
	//std::cout << "Taking Newton's Method step" << std::endl;
	//
	//float stepSize = 0.01f; // TODO: Needs to be adaptively determined

	//int numNonFixedVertices = 0;
	//std::vector<int> nonFixedIndexes; 
	//Eigen::VectorXf totalForceVector(3*mesh->coords.size());
	//totalForceVector.setZero();

	//// DENSE MATRIX CODE
	////Eigen::MatrixXf K(3*mesh->coords.size(), 3*mesh->coords.size());
	////K.setZero();
	//
	//// SPARSE MATRIX CODE
	//std::vector<Triplet> tripletsK;

	//for (int sharedCoordI = 0; sharedCoordI < mesh->coords.size(); ++sharedCoordI)
	//{
	//	if (mesh->sharedIndexBase.count(sharedCoordI) == 0)
	//	{
	//		++numNonFixedVertices;
	//		nonFixedIndexes.push_back(sharedCoordI);

	//		if (sharedCoordI >= 72)//11*11*39 )//25*16)
	//		{
	//			totalForceVector.block(3*sharedCoordI, 0, 3, 1) = totalExternalForce;  // TODO
	//		}
	//	}

	//	else 
	//	{
	//		totalForceVector.block(3*sharedCoordI, 0, 3, 1) = Eigen::Vector3f::Zero();
	//	}
	//}

	//for (int elementI = 0; elementI < mesh->elements.size(); ++elementI)
	//{
	//	HexElement * elem = (HexElement*) mesh->elements[elementI];
	//	std::vector<Eigen::Vector3f> elemDeformedCoords; 

	//	for (int ii = 0; ii < elem->vertices.size(); ++ii)
	//	{
	//		elemDeformedCoords.push_back(mesh->coords[elem->vertices[ii]]);
	//	}
	//	
	//	for (int ii = 0; ii < elem->vertices.size(); ++ii)
	//	{
	//		int sharedCoordIndex = elem->vertices[ii];
	//		
	//		if (mesh->sharedIndexBase.count(sharedCoordIndex) > 0)
	//		{
	//			continue;
	//		}

	//		Eigen::Vector3f forceOnVertex = elem->getForce(elemDeformedCoords, ii);

	//		totalForceVector.block(3*sharedCoordIndex, 0, 3, 1) = totalForceVector.block(3*sharedCoordIndex, 0, 3, 1) + forceOnVertex;
	//	}

	//	//std::cout << "Total Force Vector: " << totalForceVector << std::endl;

	//	// put element K into total K
	//	Eigen::MatrixXf elementK = elem->stiffnessMatrix(elemDeformedCoords);
	//	for (int rowI = 0; rowI < elem->vertices.size(); ++rowI) // rows
	//	{
	//		int rowSharedCoordIndex = elem->vertices[rowI];

	//		for (int colI = 0; colI < elem->vertices.size(); ++colI) // columns
	//		{
	//			int colSharedCoordIndex = elem->vertices[colI];
	//			Eigen::Matrix3f elementKBlock = elementK.block(3*rowI, 3*colI, 3, 3);
	//			
	//			// DENSE MATRIX CODE
	//			//K.block(3*rowSharedCoordIndex, 3*colSharedCoordIndex, 3, 3) += elementKBlock;

	//			// SPARSE MATRIX CODE
	//			for (int r = 0; r < 3; ++r)
	//			{
	//				for (int c = 0; c < 3; ++c)
	//				{
	//					if (elementKBlock(r,c) != 0.0f)
	//					{
	//						tripletsK.push_back( Triplet(3*rowSharedCoordIndex + r, 3*colSharedCoordIndex + c, elementKBlock(r,c)) );
	//					}
	//				}
	//			}
	//		}
	//	}
	//}

	//
	//// SPARSE MATRIC C
	//SparseMatrix K(3*mesh->coords.size(), 3*mesh->coords.size());
	//K.setFromTriplets(tripletsK.begin(), tripletsK.end());
	//tripletsK.clear();

	//Eigen::MatrixXf newK(3*numNonFixedVertices, 3*numNonFixedVertices);
	//newK.setZero();

	////cusp::array1d<int,   cusp::device_memory> I();  // row indices
 //   //cusp::array1d<int,   cusp::device_memory> J();  // column indices
 //   //cusp::array1d<float, cusp::device_memory> V();  // values

	//Eigen::VectorXf newForce = Eigen::VectorXf::Zero(3*numNonFixedVertices);

	//int nRowsNonFixed = 0;
	//for (int rowI = 0; rowI < mesh->coords.size(); ++rowI)
	//{
	//	// row fixed
	//	if (mesh->sharedIndexBase.count(rowI) > 0)
	//	{
	//		continue;
	//	}

	//	
	//	int nColsNonFixed = 0;
	//	for (int colI = 0; colI < mesh->coords.size(); ++colI)
	//	{
	//		if (mesh->sharedIndexBase.count(colI) > 0)
	//		{
	//			continue;
	//		}

	//		// DENSE MATRIX CODE
	//		// newK.block(3*nRowsNonFixed, 3*nColsNonFixed, 3, 3) += K.block(3*rowI, 3*colI, 3, 3);

	//		
	//		for (int r = 0; r < 3; ++r)
	//		{
	//			for (int c = 0; c < 3; ++c)
	//			{
	//				//I.
	//				//.push_back( Triplet(3*nRowsNonFixed + r, 3*nColsNonFixed + c, K.coeff(3*rowI + r, 3*colI + c)) );
	//			}
	//		}
	//		

	//		++nColsNonFixed;
	//	}
	//	
	//	++nRowsNonFixed;
	//}
	//
	///*
	//SparseMatrix newK(3*numNonFixedVertices, 3*numNonFixedVertices);;
	//newK.setFromTriplets(tripletsNewK.begin(), tripletsNewK.end());
	//tripletsNewK.clear();
	//*/

	//int nonFixedCount = 0;
	//for (int ii = 0; ii < mesh->coords.size(); ++ii)
	//{
	//	if (mesh->sharedIndexBase.count(ii) > 0)
	//	{
	//		continue; 
	//	}

	//	newForce.block(3*nonFixedCount, 0, 3, 1) += totalForceVector.block(3*ii, 0, 3, 1);
	//	++nonFixedCount;
	//}
	//
	////Eigen::ConjugateGradient<SparseMatrix> cg;
	////std::cout << "newK: " << newK;
	////cg.compute(newK);

	///*
	//Eigen::VectorXf deltaX  = Eigen::VectorXf::Random(3*numNonFixedVertices);
	//cg.setMaxIterations(1);
	//int i = 0;
	//do 
	//{
	//	deltaX = cg.solveWithGuess(newForce,deltaX);
	//	std::cout << i << " : " << cg.error() << std::endl;
	//	++i;
	//} while (cg.info()!=Eigen::Success && i<100);
	//*/

	////Eigen::VectorXf deltaX(3*numNonFixedVertices);
	////deltaX = cg.solve(newForce);
	////std::cout << "Error: " << cg.error() << std::endl;
	////const Eigen::VectorXf deltaX = chol.solve(newForce);

	////std::cout << "newK: " << newK << std::endl;
	////std::cout << "newForce: " << newForce << std::endl;
	////std::cout << "deltaX: " << deltaX << std::endl;

	//Eigen::VectorXf deltaX = newK.colPivHouseholderQr().solve(newForce);

	//for (int ii = 0; ii < numNonFixedVertices; ++ii)
	//{
	//	int sharedCoordIndex = nonFixedIndexes[ii];

	//	mesh->coords[sharedCoordIndex] += stepSize * deltaX.block(3*ii, 0, 3, 1);
	//}
//}
